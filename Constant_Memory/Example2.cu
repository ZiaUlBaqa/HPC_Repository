#include "hip/hip_runtime.h"
%%writefile freq_from_file.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 __constant__ int d_ConstData[100]; //constant mem in GPU
 //constant memory has to be decalred globally

__global__ void kernel(int * doubleArr)
{
    doubleArr[threadIdx.x] = d_ConstData[threadIdx.x] *  2;
}

int main()
{
  //In this example we will be utilizing constant memory of a gpu to perform some task.
  //We will be computing simple doubling of elements in an array

  int  hostData[100];
  for(int i=0; i<100; i++)
  {
    hostData[i] = i;
  }

  int * d_Data;
  hipMalloc(&d_Data,400);


  hipMemcpyToSymbol(HIP_SYMBOL(d_ConstData),hostData,400);

  kernel<<<1,100>>>(d_Data);

  hipMemcpy(hostData,d_Data,400,hipMemcpyDeviceToHost);

  for(int i = 0 ; i < 100; i++)
  {
    printf("%d ", hostData[i]);
  }

  return 0;

}