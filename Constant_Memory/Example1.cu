#include "hip/hip_runtime.h"
%%writefile freq_from_file.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 __constant__ int d_ConstData[100]; //constant mem in GPU

__global__ void kernel()
{
    printf("%d ", d_ConstData[threadIdx.x]);
}

int main()
{
  //In this example we will be utilizing constant memory of a gpu to perform some task.
  //We will be computing simple doubling of elements in an array

  int  hostData[100];
  for(int i=0; i<100; i++)
  {
    hostData[i] = i;
  }


  hipMemcpyToSymbol(HIP_SYMBOL(d_ConstData),hostData,400);

  kernel<<<1,100>>>();

  hipDeviceSynchronize();

  return 0;

}