#include "hip/hip_runtime.h"
%%writefile freq_from_file.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

// 100x10 constant array on GPU
__constant__ int d_ConstData[100][10];

__global__ void kernel(int *doubleArr)
{
    int row = blockIdx.x;
    int col = threadIdx.x;
    int idx = row * blockDim.x + col;  // flat index: 0..999

    doubleArr[idx] = d_ConstData[row][col] * 2;
}

int main()
{
    int hostData[100][10];

    // fill hostData
    for (int i = 0; i < 100; i++) {
        for (int j = 0; j < 10; j++) {
            hostData[i][j] = i;
        }
    }

    // Copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_ConstData), hostData, sizeof(hostData));

    // Allocate output array (flat)
    int *d_Data;
    hipMalloc(&d_Data, 100 * 10 * sizeof(int));

    // Launch kernel
    kernel<<<100, 10>>>(d_Data);
    hipDeviceSynchronize();

    // Copy result back
    int hostResult[100][10];
    hipMemcpy(hostResult, d_Data, sizeof(hostResult), hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < 100; i++) {
        for (int j = 0; j < 10; j++) {
            printf("%d ", hostResult[i][j]);
        }
        printf("\n");
    }

    hipFree(d_Data);
    return 0;
}
