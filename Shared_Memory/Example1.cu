#include "hip/hip_runtime.h"
%%writefile freq_from_file.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define arraysize 10000


//This was actually very slow as it was doing redundzant work by doing global -> shared ->global yaaa bakrrr whaaa????



__global__ void kernel1(int * array)
{
    __shared__ int s_array[arraysize];
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread_id < arraysize)
    {
        s_array[thread_id] = array[thread_id];
        __syncthreads();

        s_array[thread_id] += 1;

        //copying it back

        array[thread_id] = s_array[thread_id];
    }
}

__global__ void kernel2(int * array)
{

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_id < arraysize)
        array[thread_id] +=1;
}

int main()
{
    int arr[arraysize];
    for(int i=0; i<arraysize; i++)
    {
        arr[i] = i;
    }

    int * d_arr;
    hipMalloc(&d_arr,arraysize*sizeof(int));
    hipMemcpy(d_arr,arr,arraysize*sizeof(int),hipMemcpyHostToDevice);

    int * d_arr2;
    hipMalloc(&d_arr2,arraysize*sizeof(int));
    hipMemcpy(d_arr2,arr,arraysize*sizeof(int),hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel1<<<977,1024>>>(d_arr);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time: %.3f ms\n", ms);



    hipEventRecord(start);
    kernel2<<<977,1024>>>(d_arr2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time: %.3f ms\n", ms);

    return 0;
}