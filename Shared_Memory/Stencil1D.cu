#include "hip/hip_runtime.h"
%%writefile freq_from_file.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//constants
#define arraysize 1000000
#define BlockDim 1024
#define Halo 2


//This is example of stencil in 1D, ehhehehehehhe???



__global__ void kernel1(int * array)
{
    __shared__ int s_array[BlockDim + Halo];
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread_id < arraysize)
    {
        s_array[threadIdx.x+1] = array[thread_id];

        if(threadIdx.x == 0)
        {
            s_array[threadIdx.x] = s_array[threadIdx.x+1];
        }

        if(threadIdx.x == blockDim.x-1)
        {
            s_array[threadIdx.x+2] = s_array[threadIdx.x+1];
        }
        __syncthreads();

        //copying it back

        array[thread_id] = (s_array[threadIdx.x] + s_array[threadIdx.x+1] + s_array[threadIdx.x+2])/3 ;

    }
}

__global__ void kernel2(int * array)
{   int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(thread_id < arraysize)
    {
        array[thread_id] = (array[thread_id+1]+array[thread_id]+array[thread_id-1])/3;
    }
}

int main()
{
    int arr[arraysize];
    for(int i=0; i<arraysize; i++)
    {
        arr[i] = i;
    }

    int * d_arr;
    hipMalloc(&d_arr,arraysize*sizeof(int));
    hipMemcpy(d_arr,arr,arraysize*sizeof(int),hipMemcpyHostToDevice);

    int * d_arr2;
    hipMalloc(&d_arr2,arraysize*sizeof(int));
    hipMemcpy(d_arr2,arr,arraysize*sizeof(int),hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel1<<<977,1024>>>(d_arr);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time: %.3f ms\n", ms);



    hipEventRecord(start);
    kernel2<<<977,1024>>>(d_arr2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time: %.3f ms\n", ms);

    return 0;
}